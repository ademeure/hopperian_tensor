#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <hip/hip_bf16.h>
#include <assert.h>

//#define FP8

#ifdef FP8
#include <hip/hip_fp8.h>
typedef __hip_fp8_e4m3_fnuz floatX;
#define WGMMA_INSTRUCTION "wgmma.mma_async.sync.aligned.m64n256k32.f32.e4m3.e4m3"
constexpr auto CU_TENSOR_FLOATX = CU_TENSOR_MAP_DATA_TYPE_UINT8;
#define MAX_DIFF_ABS 8.0f
#define MAX_DIFF_REL 1.05f
#else
typedef __hip_bfloat16 floatX;
#define WGMMA_INSTRUCTION "wgmma.mma_async.sync.aligned.m64n256k16.f32.bf16.bf16"
constexpr auto CU_TENSOR_FLOATX = CU_TENSOR_MAP_DATA_TYPE_BFLOAT16;
#define MAX_DIFF_ABS 0.01f
#define MAX_DIFF_REL 1.001f
#endif

typedef __hip_bfloat16 floatP;
#define CUBLAS_FLOATP HIP_R_16BF
constexpr auto CU_TENSOR_FLOATP = CU_TENSOR_MAP_DATA_TYPE_BFLOAT16;

constexpr bool ENABLE_C_INPUT = false;
constexpr float ENABLE_ABSMAX_SCALING = 0.0f;
constexpr bool REDUCE_SHARED_CONFLICTS = true;

#define ENABLE_CUBLAS
#define ENABLE_RANDOM
#define ENABLE_TRUE_RANDOM
#define SLEEP_BETWEEN_KERNELS_SEC 0 // optional rest to avoid thermal throttling between kernels
constexpr bool RUN_VERIF = true;
constexpr int metadata_size = 16384; // big enough so it's spread over both L2 sides
constexpr int max_size = 16384;
constexpr int prime = 3719;
int repeat_times = 400;

int get_time() {
  static int last_time = 0;
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  int time = ts.tv_sec * 1000 + ts.tv_nsec / 1000000;
  int diff = time - last_time;
  last_time = time;
  return diff;
}

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(1);
  }
}
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

#ifdef ENABLE_RANDOM
#include <random>
std::default_random_engine generator(69);
#endif

// ...
#include "examples/matmul/matmul_10.cuh"

#ifdef ENABLE_CUBLAS
#include <hipblas.h>
hipblasHandle_t cublas_handle;
void runCublasGemmBF16(int M, int N, int K, floatP *A, floatP *B, floatP *C) {
  float alpha = 1, beta = 0;
  // C(column major) = A(row major) * B(column major)
  hipblasStatus_t status = hipblasGemmEx(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, K, &alpha, A, CUBLAS_FLOATP,
    K, B, CUBLAS_FLOATP, K, &beta, C, CUBLAS_FLOATP, M, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS error: %d\n", status);
    exit(1);
  }
}
#endif

void run_kernel(int kernel_num, int M, int N, int K, floatX *A, floatX *B, floatP *C, floatP *I, unsigned int* metadata_gpu) {
  switch (kernel_num) {
    case 0:
#ifdef ENABLE_CUBLAS
      runCublasGemmBF16(M, N, K, (floatP*)A, (floatP*)B, C);
#endif
      break;
    case 10:
      runKernel10(M, N, K, A, B, C, I, metadata_gpu);
      break;
  }
}

void randomize_matrix(floatP *mat, int N, float scale=1.0f) {
  if (scale == 0.0f) {
    for (int i = 0; i < N; i++) {
      mat[i] = (floatP)(i*1000);
    }
    return;
  }
#ifdef ENABLE_RANDOM
  std::normal_distribution<float> distribution(0, scale);
#ifdef ENABLE_TRUE_RANDOM
  for (int i = 0; i < N; i++) {
    mat[i] = (floatP)((floatX)(distribution(generator) + 0.01f));
  }
#else
  int i = 0;
  for (; i < prime; i++) {
    mat[i] = (floatP)((floatX)distribution(generator));
  }
  for (int multiplier = 1; i < N-(prime * multiplier); i += prime * multiplier, multiplier *= 2) {
    memcpy(mat+i, mat, sizeof(floatP) * prime);
  }
  for (; i < N-prime; i += prime) {
    memcpy(mat+i, mat, sizeof(floatP) * prime);
  }
  for (; i < N; i++) {
    mat[i] = mat[i-prime];
  }
#endif
#else
  hipMemset(mat, 0, sizeof(floatP) * N);
#endif
}

__global__ void verify_matrix_kernel(floatP *matRef, floatP *matOut, floatP *matI, unsigned int *error, size_t N) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N) {
    float ref_with_added = (float)((floatP)(((float)matRef[i] + (float)(ENABLE_C_INPUT ? (float)matI[i] : 0.0f))));
    float diff = fabs(ref_with_added - (float)matOut[i]);

    int x_base = i % max_size;
    int y_base = (i / max_size);

    if (diff > 0.1) {
      // (hack) accept result if it looks like RELU
      if ((float)matRef[i] > 0.0f || (float)matOut[i] != 0.0f) {
        if (diff > MAX_DIFF_ABS && ((float)ref_with_added / (float)matOut[i] > MAX_DIFF_REL || (float)ref_with_added / (float)matOut[i] < (1.0f/MAX_DIFF_REL))) {
          if(!*error) {
            printf("Divergence! Should %5.20f, Is %5.20f (Diff %5.7f) at %d\n", ref_with_added, (float)matOut[i], diff, i);
            *error = 1;
          }
        }
      }
    } else if (i < 140) {
      //: %5.20f, Is %5.20f (Diff %5.7f) at %d\n", ref_with_added, (float)matOut[i], diff, i);
    }
  }
}

__global__ void copy_to_floatX(floatP *input, floatX *output, size_t N) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N) {
    output[i] = (floatX)input[i];
  }
}

int main() {
  get_time();
  long m = max_size, n = max_size, k = max_size; // TODO: doesn't work if not all the same yet, need to debug

  floatP *A = nullptr, *B = nullptr, *C = nullptr, *I = nullptr, *C_ref = nullptr;  // host matrices
  floatP *dA = nullptr, *dB = nullptr, *dC = nullptr, *dI = nullptr, *dC_ref = nullptr; // device matrices
  floatX *dA_X = nullptr,*dB_X= nullptr, *dC_X = nullptr, *dI_X;

  A = (floatP *)malloc(sizeof(floatP) * m * k);
  B = (floatP *)malloc(sizeof(floatP) * n * k);
  C = (floatP *)malloc(sizeof(floatP) * m * n);
  I = (floatP *)malloc(sizeof(floatP) * m * n);
  C_ref = (floatP *)malloc(sizeof(floatP) * m * n);

  randomize_matrix(A, m * k);
  randomize_matrix(B, n * k);
  randomize_matrix(I, m * n, 0.0f);

  unsigned int* metadata_gpu;
  unsigned int scalar_host;
  hipMalloc((void**)&metadata_gpu, metadata_size);
  cudaCheck(hipMemset(metadata_gpu, 0, metadata_size));
  cudaCheck(hipMalloc((void **)&dA, sizeof(floatP) * m * k));
  cudaCheck(hipMalloc((void **)&dB, sizeof(floatP) * n * k));
  cudaCheck(hipMalloc((void **)&dC, sizeof(floatP) * m * n));
  cudaCheck(hipMalloc((void **)&dI, sizeof(floatP) * m * n));
  cudaCheck(hipMalloc((void **)&dC_ref, sizeof(floatP) * m * n));

  cudaCheck(hipMalloc((void **)&dA_X, sizeof(floatX) * m * k));
  cudaCheck(hipMalloc((void **)&dB_X, sizeof(floatX) * n * k));
  cudaCheck(hipMalloc((void **)&dC_X, sizeof(floatX) * m * n));
  cudaCheck(hipMalloc((void **)&dI_X, sizeof(floatX) * m * n));

  cudaCheck(hipMemcpyAsync(dA, A, sizeof(floatP) * m * k, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpyAsync(dB, B, sizeof(floatP) * n * k, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpyAsync(dC, I, sizeof(floatP) * m * n, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpyAsync(dI, I, sizeof(floatP) * m * n, hipMemcpyHostToDevice));

  copy_to_floatX<<<CEIL_DIV(m * k, 1024), 1024>>>(dA, dA_X, m * k);
  copy_to_floatX<<<CEIL_DIV(n * k, 1024), 1024>>>(dB, dB_X, n * k);
  copy_to_floatX<<<CEIL_DIV(m * n, 1024), 1024>>>(dC, dC_X, m * n);
  copy_to_floatX<<<CEIL_DIV(m * n, 1024), 1024>>>(dI, dI_X, m * n);

#ifdef ENABLE_CUBLAS
  hipblasCreate(&cublas_handle);
#endif

  timespec ts_second;
  ts_second.tv_sec = SLEEP_BETWEEN_KERNELS_SEC;
  ts_second.tv_nsec = 0;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsed_time;

  bool first_run = true;
  bool run_verif = RUN_VERIF;
  for (int kernel_num : {10, 10, 10, 10, 10, 10, 10, 0, 0}) {
    printf("\nKERNEL %d\n", kernel_num);

    if (!first_run && SLEEP_BETWEEN_KERNELS_SEC) {
      nanosleep(&ts_second, NULL); // optional rest to avoid thermal throttling between kernels
    }
    first_run = false;

#ifdef ENABLE_CUBLAS
    // Verify against cuBLAS. Also serves as a warmup step
    if (run_verif) {
      cudaCheck(hipMemset(dC, 0, sizeof(floatP) * m * n));
      cudaCheck(hipMemset(dC_ref, 0, sizeof(floatP) * m * n));
      cudaCheck(hipMemset(metadata_gpu, 0, metadata_size));

      run_kernel(kernel_num, m, n, k, dA_X, dB_X, dC, dI, metadata_gpu);
      runCublasGemmBF16(m, n, k, dA, dB, dC_ref);

      cudaCheck(hipMemset(metadata_gpu, 0, metadata_size));
      verify_matrix_kernel<<<CEIL_DIV(m * n, 1024), 1024>>>(dC_ref, dC, dI, metadata_gpu, m * n);
      hipMemcpy(&scalar_host, metadata_gpu, sizeof(unsigned int), hipMemcpyDeviceToHost); // can only be async because next memcpy isn't
      printf("\n=======> Kernel %d -> VERIFICATION: %s\n\n", kernel_num, scalar_host ? "!!!!! FAILED !!!!!" : "OK");
    }
#endif

    printf("Benchmarking kernel %d - time: %d\n", kernel_num, get_time());

    // Benchmark
    hipEventRecord(start);
    for (int j = 0; j < repeat_times; j++) {
      run_kernel(kernel_num, m, n, k, dA_X, dB_X, dC, dI, metadata_gpu);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    long flops = (2LL * m) * (n * k);
    printf( "=======> Average elapsed time: (%7.6f) s, performance: (%7.1f) TFLOPS. size: (%ld).\n\n",
        elapsed_time / 1000.0 / repeat_times, (repeat_times * flops * 1e-9) / elapsed_time, m);
    printf("Benchmarked kernel %d - time: %d\n", kernel_num, get_time());
  }

  // Free up CPU and GPU space
  free(A);
  free(B);
  free(C);
  free(I);
  free(C_ref);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  hipFree(dC_ref);
  hipFree(dA_X);
  hipFree(dB_X);
  hipFree(dC_X);
  hipFree(dI_X);
  hipFree(metadata_gpu);
  return 0;
};