#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <hip/hip_bf16.h>
//#include <hip/hip_fp8.h>
#include <assert.h>

typedef __hip_bfloat16 floatX;
typedef __hip_bfloat16 floatP;
#define CUBLAS_FLOATX HIP_R_16BF

constexpr bool ENABLE_C_INPUT = false;
constexpr float ENABLE_ABSMAX_SCALING = 0.0f;

#define ENABLE_CUBLAS
#define ENABLE_RANDOM
#define ENABLE_TRUE_RANDOM
#define SLEEP_BETWEEN_KERNELS_SEC 1 // optional rest to avoid thermal throttling between kernels
#define REFERENCE_KERNEL 0
constexpr bool RUN_VERIF = true;
constexpr int max_size = 16384;
constexpr int prime = 3719;
int repeat_times = 128;

int get_time() {
  static int last_time = 0;
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  int time = ts.tv_sec * 1000 + ts.tv_nsec / 1000000;
  int diff = time - last_time;
  last_time = time;
  return diff;
}

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(1);
  }
}
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

#ifdef ENABLE_RANDOM
#include <random>
std::default_random_engine generator(69);
#endif

// ...
#include "examples/matmul/matmul_10.cuh"

#ifdef ENABLE_CUBLAS
#include <hipblas.h>
hipblasHandle_t cublas_handle;
void runCublasGemmBF16(int M, int N, int K, floatX *A, floatX *B, floatX *C) {
  float alpha = 1, beta = 0;
  // C(column major) = A(row major) * B(column major)
  hipblasStatus_t status = hipblasGemmEx(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, K, &alpha, A, CUBLAS_FLOATX,
    N, B, CUBLAS_FLOATX, K, &beta, C, CUBLAS_FLOATX, N, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS error: %d\n", status);
    exit(1);
  }
}
#endif

void run_kernel(int kernel_num, int M, int N, int K, floatX *A, floatX *B, floatX *C, floatX *I, unsigned int* scalar_gpu) {
  switch (kernel_num) {
    case 0:
#ifdef ENABLE_CUBLAS
      runCublasGemmBF16(M, N, K, A, B, C);
#endif
      break;
    case 10:
      runKernel10(M, N, K, A, B, C, I, scalar_gpu);
      break;
  }
}

void randomize_matrix(floatX *mat, int N, float scale=1.0f) {
  if (scale == 0.0f) {
    for (int i = 0; i < N; i++) {
      mat[i] = (floatX)(i*1000);
    }
    return;
  }
#ifdef ENABLE_RANDOM
  std::normal_distribution<float> distribution(0, scale);
#ifdef ENABLE_TRUE_RANDOM
  for (int i = 0; i < N; i++) {
    mat[i] = (floatX)(distribution(generator) + 0.01f)  ;
  }
#else
  int i = 0;
  for (; i < prime; i++) {
    mat[i] = distribution(generator);
  }
  for (int multiplier = 1; i < N-(prime * multiplier); i += prime * multiplier, multiplier *= 2) {
    memcpy(mat+i, mat, sizeof(floatX) * prime);
  }
  for (; i < N-prime; i += prime) {
    memcpy(mat+i, mat, sizeof(floatX) * prime);
  }
  for (; i < N; i++) {
    mat[i] = mat[i-prime];
  }
#endif
#else
  hipMemset(mat, 0, sizeof(floatX) * N);
#endif
}

/*
bool verify_matrix(floatX *matRef, floatX *matOut, int N) {
  double diff = 0.0;
  int i;
  for (i = 0; i < N; i++) {
    int r = i / 8192, c = i % 8192;
    int it = c*8192+r;
    diff = std::fabs(__bfloat162float(matRef[i] - matOut[i]));
    if (diff > 0.1) {
      printf("Divergence! Should %5.2f, Is %5.2f (Diff %5.2f) at %d\n",
      __bfloat162float(matRef[i]), __bfloat162float(matOut[i]), diff, i);
      return false;
    }
  }
  return true;
}
*/

__global__ void verify_matrix_kernel(floatX *matRef, floatX *matOut, floatX *matI, unsigned int *error, size_t N) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;

/*
  if (i == 0) {
    for(int i = 0; i < 2048; i++) {
      float diff = fabs((float)matRef[i] - (float)matOut[i]);
      float ref = (float)matRef[i];
      float out = (float)matOut[i];
      float added = (float)matOut[i] + (float)matI[i];
      float added_ref = (float)matRef[i] + (float)matI[i];
      float value_I = (float)(matI[i]);
      printf("Should be %5.20f, Is %5.20f (Diff %5.7f) at %d (with I: %5.7f ==> real diff: %5.7f)\n", ref, out, diff, i, value_I, added_ref - out);
    }
    return;
  }
*/

  if (i < N) {
    float ref_with_added = (float)((floatX)(((float)matRef[i] + (float)(ENABLE_C_INPUT ? (float)matI[i] : 0.0f))));
    float diff = fabs(ref_with_added - (float)matOut[i]);

    int x_base = i % max_size;
    int y_base = (i / max_size);
    bool absmax_position = ENABLE_ABSMAX_SCALING ? (x_base % 256 == 0 && y_base % 256 == 0) : false;

    if (diff > 0.1 || absmax_position) {
      // accept result if it looks like RELU
      if ((float)matRef[i] > 0.0f || (float)matOut[i] != 0.0f || absmax_position) {
        //printf("Divergence! Should %5.20f, Is %5.20f (Diff %5.7f) at %d (with I: %5.7f)\n", ref_with_added, (float)matOut[i], diff, (int)i, (float)matI[i]);
        if (absmax_position) {
          // calculate absmax for the tile
          // (this is... not maximally efficient)
          float absmax = 0.0f;
          for (int x = 0; x < 256; x++) {
            for (int y = 0; y < 256; y++) {
              int idx = (x + x_base) + (y + y_base) * max_size;
              float ref_with_added = (float)((floatX)(((float)matRef[idx] + (float)(ENABLE_C_INPUT ? (float)matI[idx] : 0.0f))));
              absmax = max(absmax, fabsf(ref_with_added));
            }
          }
          floatX absmax_bf16 = (floatX)absmax;
          diff = fabsf((float)absmax_bf16 - (float)matOut[i]);
          if (diff > 2.0) {
            printf("absmax: %5.20f vs claimed_absmax: %5.20f (at: %d/%d)\n", (float)absmax_bf16, (float)matOut[i], (int)x_base, (int)y_base);
            *error = 1;
          }
        } else {
          if (diff > 100.0f && ((float)ref_with_added / (float)matOut[i] > 1.01f || (float)ref_with_added / (float)matOut[i] < 0.99f)) {
            printf("Divergence! Should %5.20f, Is %5.20f (Diff %5.7f) at %d\n", ref_with_added, (float)matOut[i], diff, i);
            *error = 1;
          }
        }
      }
    }
  }
}

int main() {
  get_time();
  long m = max_size, n = max_size, k = max_size;

  floatX *A = nullptr, *B = nullptr, *C = nullptr, *I = nullptr, *C_ref = nullptr;  // host matrices
  floatX *dA = nullptr, *dB = nullptr, *dC = nullptr, *dI = nullptr, *dC_ref = nullptr; // device matrices

  A = (floatX *)malloc(sizeof(floatX) * max_size * max_size);
  B = (floatX *)malloc(sizeof(floatX) * max_size * max_size);
  C = (floatX *)malloc(sizeof(floatX) * max_size * max_size);
  I = (floatX *)malloc(sizeof(floatX) * max_size * max_size);
  C_ref = (floatX *)malloc(sizeof(floatX) * max_size * max_size);

  randomize_matrix(A, max_size * max_size);
  randomize_matrix(B, max_size * max_size);
  randomize_matrix(I, max_size * max_size, 0.0f);

  unsigned int* scalar_gpu;
  unsigned int scalar_host;
  hipMalloc((void**)&scalar_gpu, sizeof(unsigned int));
  cudaCheck(hipMemset(scalar_gpu, 0, sizeof(unsigned int)));
  cudaCheck(hipMalloc((void **)&dA, sizeof(floatX) * max_size * max_size));
  cudaCheck(hipMalloc((void **)&dB, sizeof(floatX) * max_size * max_size));
  cudaCheck(hipMalloc((void **)&dC, sizeof(floatX) * max_size * max_size));
  cudaCheck(hipMalloc((void **)&dI, sizeof(floatX) * max_size * max_size));
  cudaCheck(hipMalloc((void **)&dC_ref, sizeof(floatX) * max_size * max_size));

  cudaCheck(hipMemcpyAsync(dA, A, sizeof(floatX) * max_size * max_size, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpyAsync(dB, B, sizeof(floatX) * max_size * max_size, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpyAsync(dC, I, sizeof(floatX) * max_size * max_size, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpyAsync(dI, I, sizeof(floatX) * max_size * max_size, hipMemcpyHostToDevice));

#ifdef ENABLE_CUBLAS
  hipblasCreate(&cublas_handle);
#endif

  timespec ts_second;
  ts_second.tv_sec = SLEEP_BETWEEN_KERNELS_SEC;
  ts_second.tv_nsec = 0;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsed_time;

  bool first_run = true;
  bool run_verif = RUN_VERIF;
  for (int kernel_num : {10}) {
    printf("\nKERNEL %d\n", kernel_num);

    if (!first_run) {
      nanosleep(&ts_second, NULL); // optional rest to avoid thermal throttling between kernels
    }
    first_run = false;

    // Verify against cuBLAS. Also serves as a warmup step
    if (run_verif) {
      cudaCheck(hipMemset(dC, 0, sizeof(floatX) * max_size * max_size));
      cudaCheck(hipMemset(dC_ref, 0, sizeof(floatX) * max_size * max_size));
      cudaCheck(hipMemset(scalar_gpu, 0, sizeof(unsigned int)));

      run_kernel(kernel_num, m, n, k, dA, dB, dC, dI, scalar_gpu);
      run_kernel(REFERENCE_KERNEL, m, n, k, dA, dB, dC_ref, dI, scalar_gpu);

      cudaCheck(hipMemset(scalar_gpu, 0, sizeof(unsigned int)));
      verify_matrix_kernel<<<CEIL_DIV(m * n, 1024), 1024>>>(dC_ref, dC, dI, scalar_gpu, m * n);
      hipMemcpy(&scalar_host, scalar_gpu, sizeof(unsigned int), hipMemcpyDeviceToHost); // can only be async because next memcpy isn't
      printf("\n=======> Kernel %d -> VERIFICATION: %s\n\n", kernel_num, scalar_host ? "!!!!! FAILED !!!!!" : "OK");
    }

    printf("Benchmarking kernel %d - time: %d\n", kernel_num, get_time());

    // Benchmark
    hipEventRecord(start);
    for (int j = 0; j < repeat_times; j++) {
      run_kernel(kernel_num, m, n, k, dA, dB, dC, dI, scalar_gpu);
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    long flops = (2LL * m) * (n * k);
    printf( "=======> Average elapsed time: (%7.6f) s, performance: (%7.1f) TFLOPS. size: (%ld).\n\n",
        elapsed_time / 1000.0 / repeat_times, (repeat_times * flops * 1e-9) / elapsed_time, m);
    printf("Benchmarked kernel %d - time: %d\n", kernel_num, get_time());
  }

  // Free up CPU and GPU space
  free(A);
  free(B);
  free(C);
  free(I);
  free(C_ref);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  hipFree(dC_ref);
  hipFree(scalar_gpu);
  return 0;
};